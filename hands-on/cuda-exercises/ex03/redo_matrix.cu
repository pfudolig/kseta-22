
#include <hip/hip_runtime.h>
#include <cassert>
#include <iostream>
#include <vector>
// Here you can set the device ID that was assigned to you
#define MYDEVICE 0

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char* msg);
// Part 2 of 4: implement the kernel
__global__ void kernel(int* a, int dimx, int dimy)
{
    int row = threadIdx.y + blockIdx.y * blockDim.y; //fill with indices of threads
    int col = threadIdx.x + blockIdx.x * blockDim.x; //same
    int val = row*dimx + col; //formula given was row*N + col
    a[val] = val; //make every value in the device pt equal to the result from the formula
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main()
{
  hipSetDevice(MYDEVICE);
  // Part 1 and 4 of 4: set the dimensions of the matrix
  int dimx = 19;
  int dimy = 67;

  std::vector<int> h_a(dimx * dimy);
  int num_bytes = dimx * dimy * sizeof(int);

  int* d_a = 0; // device and host pointers

  // allocate memory on the device
  hipMalloc(&d_a,num_bytes);

  if (NULL == d_a) {
    std::cerr << "couldn't allocate memory" << std::endl;
    return 1;
  }

  // Part 2 of 4: define grid and block size and launch the kernel
  dim3 grid, block;
  block.x = 1;
  block.y = dimy;
  grid.x  = dimx;
  grid.y  = 1;

  kernel<<<grid, block>>>(d_a, dimx, dimy);
  // block until the device has completed
  hipDeviceSynchronize();
  // check if kernel execution generated an error
  checkCUDAError("kernel execution");
  // device to host copy
  hipMemcpy(h_a.data(),d_a,num_bytes,hipMemcpyDeviceToHost);

  // Check for any CUDA errors
  checkCUDAError("hipMemcpy");
  // verify the data returned to the host is correct
  for (int row = 0; row < dimy; row++) {
    for (int col = 0; col < dimx; col++)
      assert(h_a[row * dimx + col] == row * dimx + col);
  }

  // free device memory
  hipFree(d_a);

  // If the program makes it this far, then the results are correct and
  // there are no run-time errors.  Good work!
  std::cout << "Correct!" << std::endl;

  return 0;
}

void checkCUDAError(const char* msg)
{
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    std::cerr << "Cuda error: " << msg << " " << hipGetErrorString(err)
              << std::endl;
    exit(-1);
  }
}