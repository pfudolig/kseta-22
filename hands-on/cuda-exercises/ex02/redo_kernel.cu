
#include <hip/hip_runtime.h>
#include <cassert>
#include <iostream>
#include <vector>

// Here you can set the device ID that was assigned to you
#define MYDEVICE 0

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char* msg);
 
// Part 3 of 5: implement the kernel
__global__ void myFirstKernel(int* d_a) //bc d_a is a pointer
{
  int i = threadIdx.x + blockIdx.x * blockDim.x; //global indices of threads
  d_a[i] = blockIdx.x + threadIdx.x + 42; //given
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
  hipSetDevice(MYDEVICE);

  // pointer for device memory
  int* d_a;

  // define grid and block size
  int numBlocks          = 8;
  int numThreadsPerBlock = 8;

  // host vector
  // hint: you might want to reserve some memory
  std::vector<int> h_a; //at start, array was size 0
  h_a.resize(numBlocks * numThreadsPerBlock); //host vector was empty ***set size of host vector total # of threads

  // Part 1 of 5: allocate host and device memory
  size_t memSize = numBlocks * numThreadsPerBlock * sizeof(int); //also total number of threads
  hipMalloc(&d_a, memSize); //assign memory to the location of pointer

  // Part 2 of 5: configure and launch kernel
  dim3 dimGrid(numBlocks);
  dim3 dimBlock(numThreadsPerBlock);
  myFirstKernel<<<dimGrid,dimBlock>>>(d_a);

  // block until the device has completed
  hipDeviceSynchronize();

  // check if kernel execution generated an error
  checkCUDAError("kernel execution");

  // Part 4 of 5: device to host copy
  hipMemcpy(h_a.data(),d_a,memSize,hipMemcpyDeviceToHost);

  // Check for any CUDA errors
  checkCUDAError("hipMemcpy");

  // Part 5 of 5: verify the data returned to the host is correct
  for (int i = 0; i < 8; ++i) {
    for (int j = 0; j < 8; ++j) {
     // assert(h_a[i * numThreadsPerBlock + j] == i + j + 42);
    }
  }

  // free device memory
  hipFree(d_a);

  // If the program makes it this far, then the results are correct and
  // there are no run-time errors.  Good work!
  std::cout << "Correct!" << std::endl;

  return 0;
}

void checkCUDAError(const char* msg)
{
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    std::cerr << "Cuda error: " << msg << " " << hipGetErrorString(err)
              << std::endl;
    exit(-1);
  }
}